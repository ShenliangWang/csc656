#include "hip/hip_runtime.h"
/****
File: findRedsDriver.cu
Date: 5/14/2018
By: Shenliang Wang
Compile: nvcc findRedsDriver.cu -o findreadsdriver
Run: ./findreadsdriver

****/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUMPARTICLES 32768
#define NEIGHBORHOOD .05
#define THREADSPERBLOCK 128

void initPos(float *);
float findDistance(float *, int, int);
__device__ float findDistanceGPU(float *, int, int);
void dumpResults(int index[]);

__global__ void findRedsGPU(float *p, int *numI);

int main() {
    hipEvent_t start, stop;
    float time;

    float *pos, *dpos;
    int *numReds, *dnumReds;

    pos = (float *) malloc(NUMPARTICLES * 4 * sizeof(float));
    numReds = (int *) malloc(NUMPARTICLES * sizeof(int));

    initPos(pos);

// your code to allocate device arrays for pos and numReds go here

    hipMalloc((void **)&dpos,NUMPARTICLES * 4 * sizeof(float));

    hipMalloc((void **)&dnumReds,NUMPARTICLES * sizeof(int));

    hipMemcpy(dpos,pos,NUMPARTICLES * 4 * sizeof(float),hipMemcpyHostToDevice);


// create timer events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

/* invoke kernel findRedsGPU here */

    findRedsGPU<<<NUMPARTICLES/THREADSPERBLOCK,THREADSPERBLOCK>>>(dpos,dnumReds);

    hipDeviceSynchronize();

// your code to copy results to numReds[] go here

    hipMemcpy(numReds,dnumReds,NUMPARTICLES * sizeof(int),hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Elapsed time = %f\n", time);

    dumpResults(numReds);

}

void initPos(float *p) {

// your code for initializing pos goes here
    int i;
    int j;
    for (i=0; i<NUMPARTICLES; i++) {
    p[i*4] = rand() / (float) RAND_MAX;
    p[i*4+1] = rand() / (float) RAND_MAX;
    p[i*4+2] = rand() / (float) RAND_MAX;
    j = rand() % 3;
    if (j == 0)
        p[i*4+3] = 0xff0000;
    else if (j == 1)
        p[i*4+3] = 0x00ff00;
    else
    p[i*4+3] = 0x0000ff;
}


}

__device__ float findDistanceGPU(float *p, int i, int j) {

// your code for calculating distance for particle i and j

    float x, y, z;

    x = p[i*4] - p[j*4];
    y = p[i*4+1] - p[j*4+1];
    z = p[i*4+2] - p[j*4+2];

    return(sqrt(x*x + y*y + z*z));

}

__global__ void findRedsGPU(float *p, int *numI) {

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i;
    float d;

    numI[index] = 0;
    for (i=0; i<NUMPARTICLES; i++) {
        if (index!=i) {
            d = findDistanceGPU(p, index, i);
            if (d < NEIGHBORHOOD && p[i*4+3] == 0xff0000) {
                numI[index]++;
            }
        }
    }

}
void dumpResults(int index[]) {
    int i;
    FILE *fp;

    fp = fopen("./dump.out", "w");

    for (i=0; i<NUMPARTICLES; i++) {
        fprintf(fp, "%d %d\n", i, index[i]);
    }
    fclose(fp);
}
